#include "hip/hip_runtime.h"
#include "accretion_disk.cuh"
#include "black_hole.cuh"
#include "camera.cuh"
#include "hyperparameters.cuh"

extern "C" __global__
void compute(
    float *out,
    BlackHole black_hole,
    AccretionDisk accretion_disk,
    Camera camera,
    Hyperparameters hyperparams,
    size_t numel
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numel) {
        out[i] = 0.*black_hole.radius; //sin((float) i * (float) black_hole.radius / 1800.);
    }
}
