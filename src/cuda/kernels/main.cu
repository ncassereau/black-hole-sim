#include "hip/hip_runtime.h"
#pragma diag_suppress 1444

#include <cstdio>

#include "accretion_disk.cuh"
#include "black_hole.cuh"
#include "camera.cuh"
#include "color.cuh"
#include "hyperparameters.cuh"
#include "ray.cuh"
#include "ray_tracer.cuh"
#include "skybox.cuh"

extern "C" __global__ void compute(float *output, BlackHole black_hole,
                                   AccretionDisk accretion_disk, Skybox skybox,
                                   Camera camera, Hyperparameters hyperparams) {
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px >= camera.screen_width || py >= camera.screen_height) return;

    unsigned int pixel_idx = py * camera.screen_width + px;

    float ndc_x = ((float)px + 0.5f) / camera.screen_width * 2.0f - 1.0f;
    float ndc_y = 1.0f - 2.0f * ((float)py + 0.5f) / camera.screen_height;

    Ray ray = camera.make_ray(ndc_x, ndc_y, black_hole.radius);
    Color color =
        get_ray_color(ray, black_hole, accretion_disk, skybox, hyperparams);

    output[pixel_idx * 3 + 0] = color.r;
    output[pixel_idx * 3 + 1] = color.g;
    output[pixel_idx * 3 + 2] = color.b;
}
