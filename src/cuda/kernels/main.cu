#include "hip/hip_runtime.h"
#pragma diag_suppress 1444

#include <cstdio>

#include "accretion_disk.cuh"
#include "black_hole.cuh"
#include "camera.cuh"
#include "hyperparameters.cuh"

extern "C" __global__
void compute(
    float *output,
    BlackHole black_hole,
    AccretionDisk accretion_disk,
    Camera camera,
    Hyperparameters hyperparams
) {
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px >= camera.screen_width || py >= camera.screen_height) return;

    unsigned int pixel_idx = py * camera.screen_width + px;

    float ndc_x = ((float) px + 0.5f) / camera.screen_width * 2.0f - 1.0f;
    float ndc_y = 1.0f - 2.0f * ((float) py + 0.5f) / camera.screen_height;

    Ray ray = camera.make_ray(ndc_x, ndc_y, black_hole.radius);

    printf("Ray is %f, %f, %f\n", ray.direction.x, ray.direction.y, ray.direction.z);
    output[pixel_idx * 3 + 0] = (float) ray.direction.x;
    output[pixel_idx * 3 + 1] = (float) ray.direction.y;
    output[pixel_idx * 3 + 2] = (float) ray.direction.z;
}
